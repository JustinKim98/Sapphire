// Copyright (c) 2021, Justin Kim

// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include <Motutapu/compute/cuda/Memory.hpp>
#include <hip/hip_runtime.h>

__host__ bool CudaSetDevice(int deviceId)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceId < deviceCount)
    {
        const hipError_t error = hipSetDevice(deviceId);
        return error == hipSuccess;
    }
    return false;
}

__host__ __device__ bool CudaMalloc(void** ptr, size_t bytes)
{
    const hipError_t error = hipMalloc(ptr, bytes);
    return error == hipSuccess;
}

__host__ __device__ bool CudaFree(void** ptr)
{
    const hipError_t error = hipFree(ptr);
    return error == hipSuccess;
}

__host__ bool MemcpyHostToGpu(void* gpuPtr, void* hostPtr, size_t bytes)
{
    const hipError_t error =
        hipMemcpy(gpuPtr, hostPtr, bytes, hipMemcpyHostToDevice);

    return error == hipSuccess;
}

__host__ bool MemcpyGpuToHost(void* hostPtr, void* gpuPtr, size_t bytes)
{
    const hipError_t error =
        hipMemcpy(hostPtr, gpuPtr, bytes, hipMemcpyDeviceToHost);

    return error == hipSuccess;
}
