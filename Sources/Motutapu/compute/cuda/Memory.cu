#include "hip/hip_runtime.h"
// Copyright (c) 2021, Justin Kim

// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include <Motutapu/compute/cuda/Memory.cuh>

namespace Motutapu::Compute::Cuda
{
__global__ void CopyOnGpuKernel(float* dest, const float* const src,
                                unsigned int size)
{
    const auto index = blockDim.x * blockIdx.x + threadIdx.x;

    if (index < size)
    {
        dest[index] = src[index];
    }
}

__host__ bool CudaSetDevice(int deviceId)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceId < deviceCount)
    {
        const hipError_t error = hipSetDevice(deviceId);
        return error == hipSuccess;
    }
    return false;
}

__host__ __device__ bool CudaMalloc(float** ptr, unsigned int size)
{
    const hipError_t error =
        hipMalloc(reinterpret_cast<void**>(ptr), size * sizeof(float));
    return error == hipSuccess;
}

template <typename T>
__host__ __device__ bool CudaFree(void** ptr)
{
    const hipError_t error = hipFree(ptr);
    return error == hipSuccess;
}

template <typename T>
__host__ bool MemcpyHostToGpu(T* gpuPtr, T* hostPtr, unsigned int size)
{
    const hipError_t error = hipMemcpy(
        reinterpret_cast<void*>(gpuPtr), reinterpret_cast<void*>(hostPtr),
        size * sizeof(T), hipMemcpyHostToDevice);

    return error == hipSuccess;
}

__host__ bool MemcpyHostToGpu(float* gpuPtr, float* hostPtr,
                              unsigned int size)
{
    const hipError_t error = hipMemcpy(
        reinterpret_cast<void*>(gpuPtr), reinterpret_cast<void*>(hostPtr),
        size * sizeof(float), hipMemcpyHostToDevice);

    return error == hipSuccess;
}

__host__ bool MemcpyGpuToHost(float* hostPtr, float* gpuPtr,
                              unsigned int size)
{
    const hipError_t error = hipMemcpy(
        reinterpret_cast<void*>(hostPtr), reinterpret_cast<void*>(gpuPtr),
        size * sizeof(float), hipMemcpyDeviceToHost);

    return error == hipSuccess;
}

__host__ void MemcpyGpuToGpu(float* dest, const float* src,
                             unsigned int size)
{
    unsigned int elementsCopied = 0;

    if (size > MAX_THREAD_DIM_X)
    {
        hipStream_t stream0;
        hipStreamCreate(&stream0);
        const auto requiredBlocks = size / MAX_THREAD_DIM_X;
        CopyOnGpuKernel<<<requiredBlocks, MAX_THREAD_DIM_X>>>(
            dest, src, requiredBlocks * MAX_THREAD_DIM_X);

        elementsCopied += requiredBlocks * MAX_THREAD_DIM_X;
    }

    CopyOnGpuKernel<<<1, size>>>(dest + elementsCopied,
                                 src + elementsCopied,
                                 size - elementsCopied);
}
}
