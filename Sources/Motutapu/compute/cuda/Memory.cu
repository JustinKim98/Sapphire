#include "hip/hip_runtime.h"
// Copyright (c) 2021, Justin Kim

// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include <Motutapu/compute/cuda/Memory.cuh>

namespace Motutapu::Compute::Cuda
{
__global__ void CopyOnGpuKernelBroadcast(float* dst, const float* const src,
                                         unsigned int srcStride,
                                         unsigned int size)
{
    const auto sizePerBlock = size / gridDim.x;
    const auto numLoops = sizePerBlock / blockDim.x;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        dst[blockOffset + blockDim.x * i + threadIdx.x] =
            src[(blockOffset + blockDim.x * i + threadIdx.x) % srcStride];
    }
}

__global__ void CopyOnGpuKernel(float* dst, const float* const src,
                                unsigned int size)
{
    const auto sizePerBlock = size / gridDim.x;
    const auto numLoops = sizePerBlock / blockDim.x;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        dst[blockOffset + blockDim.x * i + threadIdx.x] =
            src[blockOffset + blockDim.x * i + threadIdx.x];
    }
}

__host__ bool CudaSetDevice(int deviceId)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceId < deviceCount)
    {
        const hipError_t error = hipSetDevice(deviceId);
        return error == hipSuccess;
    }
    return false;
}

__host__ __device__ bool CudaMalloc(void** ptr, unsigned int size)
{
    const hipError_t error = hipMalloc((void**)ptr, size);
    return error == hipSuccess;
}

__host__ __device__ bool CudaFree(void* ptr)
{
    const hipError_t error = hipFree((void*)(ptr));
    return error == hipSuccess;
}

__host__ bool CopyHostToGpu(void* gpuPtr, void* hostPtr, unsigned int size)
{
    const hipError_t error = hipMemcpy((void*)(gpuPtr), (void*)(hostPtr),
                                         size, hipMemcpyHostToDevice);

    return error == hipSuccess;
}

__host__ bool CopyGpuToHost(void* hostPtr, void* gpuPtr, unsigned int size)
{
    const hipError_t error = hipMemcpy((void*)(hostPtr), (void*)(gpuPtr),
                                         size, hipMemcpyDeviceToHost);

    return error == hipSuccess;
}

__host__ bool CopyGpuToGpu(void* dst, const void* src, unsigned int byteSize)
{
    const hipError_t error =
        hipMemcpy(dst, src, byteSize, hipMemcpyDeviceToDevice);
    return error == hipSuccess;
}

__host__ bool CopyGpuToGpuAsync(float* dst, const float* src,
                                unsigned int byteSize, hipStream_t stream)
{
    const hipError_t error =
        hipMemcpyAsync(dst, src, byteSize, hipMemcpyDeviceToDevice, stream);
    return error == hipSuccess;
}

__host__ bool CopyGpuToGpuBroadcast(float* dst, const float* src,
                                    unsigned int byteSize,
                                    unsigned int srcStrideByteSize)
{
    for (unsigned int idx = 0; idx < byteSize; idx += srcStrideByteSize)
    {
        const hipError_t error = hipMemcpy(dst + idx, src, srcStrideByteSize,
                                             hipMemcpyDeviceToDevice);
        if (error != hipSuccess)
            return false;
    }

    return true;
}

}  // namespace Motutapu::Compute::Cuda
