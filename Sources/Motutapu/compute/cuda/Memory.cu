#include "hip/hip_runtime.h"
// Copyright (c) 2021, Justin Kim

// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include <Motutapu/compute/cuda/Memory.cuh>

namespace Motutapu::Compute::Cuda
{
__global__ void CopyOnGpuKernelBroadcast(float* dst, const float* const src,
                                         unsigned int srcStride,
                                         unsigned int size)
{
    const auto sizePerBlock = size / gridDim.x;
    const auto numLoops = sizePerBlock / blockDim.x;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        dst[blockOffset + blockDim.x * i + threadIdx.x] =
            src[(blockOffset + blockDim.x * i + threadIdx.x) % srcStride];
    }
}

__global__ void CopyOnGpuKernel(float* dst, const float* const src,
                                unsigned int size)
{
    const auto sizePerBlock = size / gridDim.x;
    const auto numLoops = sizePerBlock / blockDim.x;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        dst[blockOffset + blockDim.x * i + threadIdx.x] =
            src[blockOffset + blockDim.x * i + threadIdx.x];
    }
}

__host__ bool CudaSetDevice(int deviceId)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceId < deviceCount)
    {
        const hipError_t error = hipSetDevice(deviceId);
        return error == hipSuccess;
    }
    return false;
}

__host__ __device__ bool CudaMalloc(void** ptr, unsigned int size)
{
    const hipError_t error = hipMalloc((void**)ptr, size);
    return error == hipSuccess;
}

__host__ __device__ bool CudaFree(void* ptr)
{
    const hipError_t error = hipFree((void*)(ptr));
    return error == hipSuccess;
}

__host__ bool CopyHostToGpu(void* gpuPtr, void* hostPtr, unsigned int size)
{
    const hipError_t error = hipMemcpy((void*)(gpuPtr), (void*)(hostPtr),
                                         size, hipMemcpyHostToDevice);

    return error == hipSuccess;
}

__host__ bool CopyGpuToHost(void* hostPtr, void* gpuPtr, unsigned int size)
{
    const hipError_t error = hipMemcpy((void*)(hostPtr), (void*)(gpuPtr),
                                         size, hipMemcpyDeviceToHost);

    return error == hipSuccess;
}

__host__ void CopyGpuToGpu(void* dst, const void* src, unsigned int size)
{
    const auto numLoops = 16;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto blockDim = size / (threadDim * numLoops);
    const auto firstLaunchSize = blockDim * threadDim * numLoops;

    if (firstLaunchSize > 0)
        CopyOnGpuKernel<<<blockDim, threadDim>>>(dst, src, firstLaunchSize);
    if (size > firstLaunchSize)
        CopyOnGpuKernel<<<1, size - firstLaunchSize>>>(dst + firstLaunchSize,
                                                       src + firstLaunchSize,
                                                       size - firstLaunchSize);
}

__host__ void CopyGpuToGpuAsync(float* dst, const float* src, unsigned int size,
                                hipStream_t stream)
{
    const auto numLoops = 16;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto blockDim = size / (threadDim * numLoops);
    const auto firstLaunchSize = blockDim * threadDim * numLoops;

    if (firstLaunchSize > 0)
        CopyOnGpuKernel<<<blockDim, threadDim, 0, stream>>>(dst, src,
                                                            firstLaunchSize);
    hipStreamSynchronize(stream);

    if (size > firstLaunchSize)
        CopyOnGpuKernel<<<1, size - firstLaunchSize, 0, stream>>>(
            dst + firstLaunchSize, src + firstLaunchSize,
            size - firstLaunchSize);
}

__host__ void CopyGpuToGpuBroadcast(float* dst, const float* src,
                                    unsigned int size, unsigned int srcStride)
{
    const auto numLoops = 8;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto blockDim = size / (threadDim * numLoops);
    const auto firstLaunchSize = blockDim * threadDim * numLoops;

    if (firstLaunchSize > 0)
        CopyOnGpuKernelBroadcast<<<blockDim, threadDim>>>(dst, src, srcStride,
                                                          firstLaunchSize);
    if (size > firstLaunchSize)
        CopyOnGpuKernelBroadcast<<<1, size - firstLaunchSize>>>(
            dst + firstLaunchSize, src, srcStride, size - firstLaunchSize);
}

}  // namespace Motutapu::Compute::Cuda
