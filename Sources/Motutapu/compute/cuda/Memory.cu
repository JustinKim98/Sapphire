// Copyright (c) 2021, Justin Kim

// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include <Motutapu/compute/cuda/Memory.cuh>

namespace Motutapu::Compute::Cuda
{
__host__ bool CudaSetDevice(int deviceId)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceId < deviceCount)
    {
        const hipError_t error = hipSetDevice(deviceId);
        return error == hipSuccess;
    }
    return false;
}

__host__ __device__ bool CudaMallocFloat(float** ptr, unsigned int size)
{
    const hipError_t error =
        hipMalloc(reinterpret_cast<void**>(ptr), size * sizeof(float));
    return error == hipSuccess;
}

__host__ __device__ bool CudaMallocHalf(half** ptr, unsigned int size)
{
    const hipError_t error =
        hipMalloc(reinterpret_cast<void**>(ptr), size * sizeof(float));
    return error == hipSuccess;
}

template <typename T>
__host__ __device__ bool CudaFree(void** ptr)
{
    const hipError_t error = hipFree(ptr);
    return error == hipSuccess;
}

template <typename T>
__host__ bool MemcpyHostToGpu(T* gpuPtr, T* hostPtr, unsigned int size)
{
    const hipError_t error = hipMemcpy(
        reinterpret_cast<void*>(gpuPtr), reinterpret_cast<void*>(hostPtr),
        size * sizeof(T), hipMemcpyHostToDevice);

    return error == hipSuccess;
}

__host__ bool MemcpyHostToGpuFloat(float* gpuPtr, float* hostPtr, unsigned int size)
{
    const hipError_t error = hipMemcpy(
        reinterpret_cast<void*>(gpuPtr), reinterpret_cast<void*>(hostPtr),
        size * sizeof(float), hipMemcpyHostToDevice);

    return error == hipSuccess;
}

__host__ bool MemcpyHostToGpuHalf(half* gpuPtr, half* hostPtr,
                                   unsigned int size)
{
    const hipError_t error = hipMemcpy(
        reinterpret_cast<void*>(gpuPtr), reinterpret_cast<void*>(hostPtr),
        size * sizeof(half), hipMemcpyHostToDevice);

    return error == hipSuccess;
}

__host__ bool MemcpyGpuToHostFloat(float* hostPtr, float* gpuPtr,
                                   unsigned int size)
{
    const hipError_t error = hipMemcpy(
        reinterpret_cast<void*>(hostPtr), reinterpret_cast<void*>(gpuPtr),
        size * sizeof(float), hipMemcpyDeviceToHost);

    return error == hipSuccess;
}

__host__ bool MemcpyGpuToHostHalf(half* hostPtr, half* gpuPtr,
                                   unsigned int size)
{
    const hipError_t error = hipMemcpy(
        reinterpret_cast<void*>(hostPtr), reinterpret_cast<void*>(gpuPtr),
        size * sizeof(half), hipMemcpyDeviceToHost);

    return error == hipSuccess;
}

__host__ void MemcpyGpuToGpuFloat(float* dest, const float* src, unsigned int size)
{
    unsigned int elementsCopied = 0;

    if (size > MAX_THREAD_DIM_X)
    {
        hipStream_t stream0;
        hipStreamCreate(&stream0);
        const auto requiredBlocks = size / MAX_THREAD_DIM_X;
        CopyOnGpu<float><<<requiredBlocks, MAX_THREAD_DIM_X>>>(
            dest, src, requiredBlocks * MAX_THREAD_DIM_X);

        elementsCopied += requiredBlocks * MAX_THREAD_DIM_X;
    }

    CopyOnGpu<float><<<1, size>>>(dest + elementsCopied, src + elementsCopied,
                                  size - elementsCopied);
}

__host__ void MemcpyGpuToGpuHalf(half* dest, const half* src,
                                 unsigned int size)
{
    unsigned int elementsCopied = 0;

    if (size > MAX_THREAD_DIM_X)
    {
        hipStream_t stream0;
        hipStreamCreate(&stream0);
        const auto requiredBlocks = size / MAX_THREAD_DIM_X;
        CopyOnGpu<half><<<requiredBlocks, MAX_THREAD_DIM_X>>>(
            dest, src, requiredBlocks * MAX_THREAD_DIM_X);

        elementsCopied += requiredBlocks * MAX_THREAD_DIM_X;
    }

    CopyOnGpu<half><<<1, size>>>(dest + elementsCopied, src + elementsCopied,
                                  size - elementsCopied);
}
}
