#include "hip/hip_runtime.h"
// Copyright (c) 2021, Justin Kim

// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.


#include <Motutapu/compute/cuda/dense/DenseMatmul.hpp>
#include <mma.h>

#define WARP_SIZE 32

#if SHARED_MEMORY_LIMIT_64K
#define CHUNK_K 4
#else
#define CHUNK_K 8
#endif

namespace Motutapu::Cuda::Dense
{

using namespace nvcuda;

//! Matrix is divided into chunk with (chunkSize * chunkSize) tiles/
//! Each chunk has tile, which is (16 x 16)
//! Each warp computes each tile as output
//! for 64KB shared memory, use chunkSize of 4
//! We use chunkSize x chunkSize warps in total
//! Each warp is responsible for 16 x 16 tile in output matrix

//! Each chunk is composed of 16*16 elements
//! in case of float(fp32), each matrix tile will take
//! (16 x 16) x (chunkSize chunkSize) x 4;
//! in case of half(fp16), each matrix tile will take
//! (16 x 16) x (chunkSize x chunkSize) x 2 = 8 KB
//! se use chunkSize of 4 for half considering shift
//! we reserve some space on shared memory for shift space to protect from
//! bank conflicts
//! for shared memory more than 64KB, we use chunkSize of 4
//! for float(fp32) data types
//! and we use chunkSize of 8 for half(fp16) data types

//! Each warp will pick each row of A, and column of B
//! and perform fma(fused multiply add) with iterations
//! Iteration number will be same as chunk size

//! We assume A, B, and Out is in row-wise, and 256bit aligned
//! chunkSize must be 4 if shared memory is equal or less than 64kB
//! otherwise it can be set 8
__global__ void WmmaGemmHalf(half* Out, half* A, half* B, size_t numRowOutA,
                             size_t numRowOutB, size_t numColARowB,
                             size_t paddedColSizeA, size_t paddedColSizeB,
                             size_t paddedColSizeOut, size_t size)
{
    constexpr size_t tileDim = 16;
    constexpr size_t chunkSize = CHUNK_K;
    // Minimum shift we can use with 256bit alignment while protecting from bank
    // conflicts;
    constexpr size_t shift = 32 / sizeof(half);
    constexpr size_t shiftedSharedMemoryColSize = chunkSize * tileDim + shift;

    //! chunkSize*tileDim is 32 or 64 depending ong size of
    extern __shared__ half sharedMemory[][shiftedSharedMemoryColSize];

    const size_t chunkIdxM = blockIdx.x;
    const size_t chunkIdxN = blockIdx.y;
    const size_t chunkIdxK = blockIdx.z;

    const size_t warpIdx = threadIdx.x / WARP_SIZE;
    const size_t laneIdx = threadIdx.x % WARP_SIZE;

    const size_t tileRowIdx = warpIdx / 4;
    const size_t tileColIdx = warpIdx % 4;

    const half* chunkPtrA = A + paddedColSizeA * chunkIdxM * tileDim * chunkSize
                            +
                            chunkIdxK * tileDim * chunkSize;
    const half* chunkPtrB = B + paddedColSizeB * chunkIdxK * tileDim * chunkSize
                            +
                            chunkIdxN * tileDim * chunkSize;
    half* chunkPtrOut = Out + paddedColSizeOut * chunkIdxM * tileDim * chunkSize
                        +
                        chunkIdxN * tileDim * chunkSize;

    const half* tilePtrA = chunkPtrA + paddedColSizeA * tileRowIdx * tileDim;
    const half* tilePtrB = chunkPtrB + tileColIdx * tileDim;
    half* tilePtrOut = chunkPtrOut + paddedColSizeOut * tileRowIdx * tileDim +
                       tileColIdx * tileDim;

    const size_t matrixBOffset = chunkSize * tileDim;

    //! For half of the warps, copy matrix A while other half copies B
    const half* copyPtr;

    size_t sharedMemCopyRowIdx;
    if (laneIdx % 2)
    {
        copyPtr = tilePtrA + paddedColSizeA * (laneIdx / 2);
        sharedMemCopyRowIdx = tileRowIdx + laneIdx / 2;
    }
    else
    {
        copyPtr = tilePtrB + paddedColSizeB * (laneIdx / 2);
        sharedMemCopyRowIdx = matrixBOffset + tileRowIdx + laneIdx / 2;
    }

    //! Load the matrix to shared memory
    //! each thread copies consecutive row from their src determined previously
#pragma unroll
    for (int i = 0; i < tileDim; i++)
    {
        const size_t sharedMemCopyColIdx = tileColIdx * tileDim + i;
        sharedMemory[sharedMemCopyRowIdx][sharedMemCopyColIdx + i] =
            *(copyPtr + i);
    }

    //! Load shared memory to fragments and accumulate
    wmma::fragment<wmma::matrix_a, tileDim, tileDim, tileDim, half,
                   wmma::row_major>
        fragA;
    wmma::fragment<wmma::matrix_b, tileDim, tileDim, tileDim, half,
                   wmma::row_major>
        fragB;
    wmma::fragment<wmma::accumulator, tileDim, tileDim, tileDim, half> fragAcc;
    wmma::fragment<wmma::accumulator, tileDim, tileDim, tileDim, half> fragOut;

    wmma::fill_fragment(fragAcc, 0.0f);

    for (int i = 0; i < chunkSize; ++i)
    {
        wmma::load_matrix_sync(fragA, &sharedMemory[tileRowIdx][tileDim * i],
                               shiftedSharedMemoryColSize);
        wmma::load_matrix_sync(
            fragB, &sharedMemory[tileDim * i + matrixBOffset][tileColIdx],
            shiftedSharedMemoryColSize);
        wmma::mma_sync(fragAcc, fragA, fragB, fragAcc);
    }

    wmma::load_matrix_sync(fragOut, tilePtrOut, paddedColSizeOut,
                           wmma::mem_row_major);

    wmma::store_matrix_sync(tilePtrOut, fragAcc, paddedColSizeOut,
                            wmma::mem_row_major);
}

__global__ void WmmaGemmFloat(float* Out, half* A, half* B, size_t numRowOutA,
                              size_t numRowOutB, size_t numColARowB,
                              size_t paddedColSizeA, size_t paddedColSizeB,
                              size_t paddedColSizeOut, size_t size)
{
    constexpr size_t tileDim = 16;
    constexpr size_t chunkSize = CHUNK_K;
    // Minimum shift we can use with 256bit alignment while protecting from bank
    // conflicts;
    constexpr size_t shift = 32 / sizeof(float);
    constexpr size_t shiftedSharedMemoryColSize = chunkSize * tileDim + shift;

    //! chunkSize*tileDim is 32 or 64 depending ong size of
    extern __shared__ half sharedMemory[][shiftedSharedMemoryColSize];

    const size_t tileIdxM = blockIdx.x;
    const size_t tileIdxN = blockIdx.y;
    const size_t tileIdxK = blockIdx.z;

    const size_t warpIdx = threadIdx.x / WARP_SIZE;
    const size_t laneIdx = threadIdx.x % WARP_SIZE;

    const size_t tileRowIdx = warpIdx / 4;
    const size_t tileColIdx = warpIdx % 4;

    const half* chunkPtrA = A + paddedColSizeA * tileIdxM * tileDim * chunkSize
                             +
                             tileIdxK * tileDim * chunkSize;
    const half* chunkPtrB = B + paddedColSizeB * tileIdxK * tileDim * chunkSize
                             +
                             tileIdxN * tileDim * chunkSize;
    float* chunkPtrOut = Out + paddedColSizeOut * tileIdxM * tileDim * chunkSize
                         +
                         tileIdxN * tileDim * chunkSize;

    const half* tilePtrA = chunkPtrA + paddedColSizeA * tileRowIdx * tileDim;
    const half* tilePtrB = chunkPtrB + tileColIdx * tileDim;
    float* tilePtrOut = chunkPtrOut + paddedColSizeOut * tileRowIdx * tileDim +
                        tileColIdx * tileDim;

    const size_t matrixBOffset = chunkSize * tileDim;

    //! For half of the warps, copy matrix A while other half copies B
    const half* copyPtr;

    size_t sharedMemCopyRowIdx;
    if (laneIdx % 2)
    {
        copyPtr = tilePtrA + paddedColSizeA * (laneIdx / 2);
        sharedMemCopyRowIdx = tileRowIdx + laneIdx / 2;
    }
    else
    {
        copyPtr = tilePtrB + paddedColSizeB * (laneIdx / 2);
        sharedMemCopyRowIdx = matrixBOffset + tileRowIdx + laneIdx / 2;
    }

    //! Load the matrix to shared memory
    //! each thread copies consecutive row from their src determined previously
#pragma unroll
    for (int i = 0; i < tileDim; i++)
    {
        const size_t sharedMemCopyColIdx = tileColIdx * tileDim + i;
        sharedMemory[sharedMemCopyRowIdx][sharedMemCopyColIdx + i] =
            *(copyPtr + i);
    }

    //! Load shared memory to fragments and accumulate
    wmma::fragment<wmma::matrix_a, tileDim, tileDim, tileDim, half,
                   wmma::row_major>
        fragA;
    wmma::fragment<wmma::matrix_b, tileDim, tileDim, tileDim, half,
                   wmma::row_major>
        fragB;

    wmma::fragment<wmma::accumulator, tileDim, tileDim, tileDim, float> fragAcc;
    wmma::fragment<wmma::accumulator, tileDim, tileDim, tileDim, float> fragOut;

    wmma::fill_fragment(fragAcc, 0.0f);

    for (int i = 0; i < chunkSize; ++i)
    {
        wmma::load_matrix_sync(fragA, &sharedMemory[tileRowIdx][tileDim * i],
                               shiftedSharedMemoryColSize);
        wmma::load_matrix_sync(
            fragB, &sharedMemory[tileDim * i + matrixBOffset][tileColIdx],
            shiftedSharedMemoryColSize);
        wmma::mma_sync(fragAcc, fragA, fragB, fragAcc);
    }

    wmma::load_matrix_sync(fragOut, tilePtrOut, paddedColSizeOut,
                           wmma::mem_row_major);

    wmma::store_matrix_sync(tilePtrOut, fragAcc, paddedColSizeOut,
                            wmma::mem_row_major);
}
}
