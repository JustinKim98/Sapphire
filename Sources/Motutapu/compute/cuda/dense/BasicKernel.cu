#include "hip/hip_runtime.h"
// Copyright (c) 2021, Justin Kim

// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include <Motutapu/compute/cuda/dense/BasicKernel.cuh>

namespace Motutapu::Compute::Cuda::Dense
{
//__global__ void transposeNaive(float* output, const float* input,
//                               unsigned int inputNumRows,
//                               unsigned int inputNumCols, bool broadcastInput)
//{
//    int x = blockIdx.x * TILE_DIM + threadIdx.x;
//    int y = blockIdx.y * TILE_DIM + threadIdx.y;
//    int width = gridDim.x * TILE_DIM;
//
//    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
//        output[x * width + (y + j)] = input[(y + j) * width + x];
//}

//! (x,y) : (TILE_DIM*8) threads per block
//! Assuming input is M x N, (nx, ny, nz) : (N/TILE_DIM, M/TILE_DIM, batchSize)
//! blocks required
__global__ void TransposeKernel(float* output, const float* input,
                                unsigned int inputNumRows,
                                unsigned int inputNumCols, bool broadcastInput)
{
    __shared__ float tile[TILE_DIM][TILE_DIM + 1];

    const unsigned int outputNumRows = inputNumCols;
    const unsigned int outputNumCols = inputNumRows;

    const int inputColIdx = blockIdx.x * TILE_DIM + threadIdx.x;
    const int inputRowIdx = blockIdx.y * TILE_DIM + threadIdx.y;

    const int outputColIdx = blockIdx.y * TILE_DIM + threadIdx.x;
    const int outputRowIdx = blockIdx.x * TILE_DIM + threadIdx.y;

    float* outputOffset = output + inputNumRows * inputNumCols * blockIdx.z;
    const float* inputOffset =
        input + (broadcastInput ? 0 : inputNumRows * inputNumCols * blockIdx.z);

    for (int i = 0; (i < TILE_DIM) && (inputRowIdx * i < inputNumRows); i += 8)
    {
        if (inputRowIdx + i < inputNumRows && inputColIdx < inputNumCols)
            tile[threadIdx.y + i][threadIdx.x] =
                inputOffset[(inputRowIdx + i) * inputNumCols + inputColIdx];
    }

    __syncthreads();

    for (int i = 0; (i < TILE_DIM) && (outputRowIdx * i < outputNumRows);
         i += 8)
    {
        if (outputRowIdx + i < outputNumRows && outputColIdx < outputNumCols)
            outputOffset[(outputRowIdx + i) * outputNumCols + outputColIdx] =
                tile[threadIdx.x][threadIdx.y + i];
    }
}

__global__ void AddKernel(float* output, const float* inputA,
                          const float* inputB, unsigned int offset,
                          unsigned int launchSize, unsigned int totalSize,
                          unsigned int inputStride, bool broadcastInputA,
                          bool broadcastInputB)
{
    const auto sizePerBlock = launchSize / gridDim.x;
    const auto numLoops = sizePerBlock / blockDim.x;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    unsigned int leftOverA = broadcastInputA ? inputStride : totalSize;
    unsigned int leftOverB = broadcastInputB ? inputStride : totalSize;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        output[offset + blockOffset + blockDim.x * i + threadIdx.x] =
            inputA[(offset + blockOffset + blockDim.x * i + threadIdx.x) %
                   leftOverA] +
            inputB[(offset + blockOffset + blockDim.x * i + threadIdx.x) %
                   leftOverB];
    }
}

__global__ void AddKernelShared(float* output, const float* inputA,
                                const float* inputB, unsigned int offset,
                                unsigned int launchSize, unsigned int totalSize,
                                unsigned int inputStride, unsigned int numLoops,
                                bool broadcastInputA, bool broadcastInputB)
{
    __shared__ extern float temp[];

    const auto sizePerBlock = blockDim.x * numLoops;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    unsigned int leftOverA = broadcastInputA ? inputStride : totalSize;
    unsigned int leftOverB = broadcastInputB ? inputStride : totalSize;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        if (offset + blockOffset + blockDim.x * i + threadIdx.x < totalSize)
        {
            temp[blockDim.x * i + threadIdx.x] =
                inputA[(offset + blockOffset + blockDim.x * i + threadIdx.x) %
                       leftOverA];

            temp[sizePerBlock + blockDim.x * i + threadIdx.x] =
                inputB[(offset + blockOffset + blockDim.x * i + threadIdx.x) %
                       leftOverB];
        }
    }

    __syncthreads();

    for (unsigned int i = 0; i < numLoops; i++)
    {
        if (offset + blockOffset + blockDim.x * i + threadIdx.x < totalSize)
        {
            output[offset + blockOffset + blockDim.x * i + threadIdx.x] =
                temp[blockDim.x * i + threadIdx.x] +
                temp[sizePerBlock + blockDim.x * i + threadIdx.x];
        }
    }
}

__global__ void AddKernelBroadcast(float* output, const float* inputA,
                                   const float* inputB, unsigned int offset,
                                   unsigned int totalSize,
                                   unsigned int inputStride,
                                   bool broadcastInputA, bool broadcastInputB)
{
    __shared__ extern float temp[];
    const auto id = blockDim.x * blockIdx.x + threadIdx.x;

    if (id < inputStride)
    {
        if (broadcastInputA)
            temp[id % blockDim.x] = inputA[offset + id];

        if (broadcastInputB)
            temp[blockDim.x + id % blockDim.x] = inputB[offset + id];

        __syncthreads();

        for (int i = 0; i < totalSize; i += inputStride)
        {
            auto aValue = broadcastInputA ? temp[id % blockDim.x]
                                          : inputA[offset + id + i];
            auto bValue = broadcastInputB ? temp[blockDim.x + id % blockDim.x]
                                          : inputB[offset + id + i];
            output[offset + id + i] = aValue + bValue;
        }
    }
}

__global__ void SubKernel(float* output, const float* inputA,
                          const float* inputB, unsigned int offset,
                          unsigned int launchSize, unsigned int totalSize,
                          unsigned int inputStride, bool broadcastInputA,
                          bool broadcastInputB)
{
    const auto sizePerBlock = launchSize / gridDim.x;
    const auto numLoops = sizePerBlock / blockDim.x;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    unsigned int leftOverA = broadcastInputA ? inputStride : totalSize;
    unsigned int leftOverB = broadcastInputB ? inputStride : totalSize;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        output[offset + blockOffset + blockDim.x * i + threadIdx.x] =
            inputA[(offset + blockOffset + blockDim.x * i + threadIdx.x) %
                   leftOverA] -
            inputB[(offset + blockOffset + blockDim.x * i + threadIdx.x) %
                   leftOverB];
    }
}

__global__ void DotKernel(float* output, const float* inputA,
                          const float* inputB, unsigned int offset,
                          unsigned int launchSize, unsigned int totalSize,
                          unsigned int inputStride, bool broadcastInputA,
                          bool broadcastInputB)
{
    const auto sizePerBlock = launchSize / gridDim.x;
    const auto numLoops = sizePerBlock / blockDim.x;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    unsigned int leftOverA = broadcastInputA ? inputStride : totalSize;
    unsigned int leftOverB = broadcastInputB ? inputStride : totalSize;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        output[offset + blockOffset + blockDim.x * i + threadIdx.x] =
            inputA[(offset + blockOffset + blockDim.x * i + threadIdx.x) %
                   leftOverA] *
            inputB[(offset + blockOffset + blockDim.x * i + threadIdx.x) %
                   leftOverB];
    }
}

__global__ void ScaleKernel(float* output, const float* input,
                            const float scaleFactor, unsigned int totalSize)
{
    const auto sizePerBlock = totalSize / gridDim.x;
    const auto numLoops = sizePerBlock / blockDim.x;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        output[blockOffset + blockDim.x * i + threadIdx.x] =
            input[blockOffset + blockDim.x * i + threadIdx.x] * scaleFactor;
    }
}

__global__ void PowKernel(float* output, const float* input, const float factor,
                          unsigned int totalSize)
{
    const auto sizePerBlock = totalSize / gridDim.x;
    const auto numLoops = sizePerBlock / blockDim.x;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        output[blockOffset + blockDim.x * i + threadIdx.x] =
            powf(input[blockOffset + blockDim.x * i + threadIdx.x], factor);
    }
}

__global__ void cosKernel(float* output, const float* input,
                          unsigned int totalSize)
{
    const auto sizePerBlock = totalSize / gridDim.x;
    const auto numLoops = sizePerBlock / blockDim.x;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        output[blockOffset + blockDim.x * i + threadIdx.x] =
            cosf(input[blockOffset + blockDim.x * i + threadIdx.x]);
    }
}

__global__ void sinKernel(float* output, const float* input,
                          unsigned int totalSize)
{
    const auto sizePerBlock = totalSize / gridDim.x;
    const auto numLoops = sizePerBlock / blockDim.x;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        output[blockOffset + blockDim.x * i + threadIdx.x] =
            sinf(input[blockOffset + blockDim.x * i + threadIdx.x]);
    }
}

__global__ void tanKernel(float* output, const float* input,
                          unsigned int totalSize)
{
    const auto sizePerBlock = totalSize / gridDim.x;
    const auto numLoops = sizePerBlock / blockDim.x;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        output[blockOffset + blockDim.x * i + threadIdx.x] =
            tanf(input[blockOffset + blockDim.x * i + threadIdx.x]);
    }
}

__global__ void coshKernel(float* output, const float* input,
                           unsigned int totalSize)
{
    const auto sizePerBlock = totalSize / gridDim.x;
    const auto numLoops = sizePerBlock / blockDim.x;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        output[blockOffset + blockDim.x * i + threadIdx.x] =
            coshf(input[blockOffset + blockDim.x * i + threadIdx.x]);
    }
}

__global__ void sinhKernel(float* output, const float* input,
                           unsigned int totalSize)
{
    const auto sizePerBlock = totalSize / gridDim.x;
    const auto numLoops = sizePerBlock / blockDim.x;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        output[blockOffset + blockDim.x * i + threadIdx.x] =
            sinhf(input[blockOffset + blockDim.x * i + threadIdx.x]);
    }
}

__global__ void tanhKernel(float* output, const float* input,
                           unsigned int totalSize)
{
    const auto sizePerBlock = totalSize / gridDim.x;
    const auto numLoops = sizePerBlock / blockDim.x;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        output[blockOffset + blockDim.x * i + threadIdx.x] =
            tanhf(input[blockOffset + blockDim.x * i + threadIdx.x]);
    }
}

__global__ void logKernel(float* output, const float* input,
                          unsigned int totalSize)
{
    const auto sizePerBlock = totalSize / gridDim.x;
    const auto numLoops = sizePerBlock / blockDim.x;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        output[blockOffset + blockDim.x * i + threadIdx.x] =
            logf(input[blockOffset + blockDim.x * i + threadIdx.x]);
    }
}

__global__ void log10Kernel(float* output, const float* input,
                            unsigned int totalSize)
{
    const auto sizePerBlock = totalSize / gridDim.x;
    const auto numLoops = sizePerBlock / blockDim.x;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        output[blockOffset + blockDim.x * i + threadIdx.x] =
            log10f(input[blockOffset + blockDim.x * i + threadIdx.x]);
    }
}

__global__ void ReLUKernel(float* output, const float* input,
                           unsigned int totalSize)
{
    const auto sizePerBlock = totalSize / gridDim.x;
    const auto numLoops = sizePerBlock / blockDim.x;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        output[blockOffset + blockDim.x * i + threadIdx.x] =
            input[blockOffset + blockDim.x * i + threadIdx.x] > 0.0f
                ? input[blockOffset + blockDim.x * i + threadIdx.x]
                : 0.0f;
    }
}

__global__ void ReLUDerivativeKernel(float* output, const float* input,
                                     unsigned int totalSize)
{
    const auto sizePerBlock = totalSize / gridDim.x;
    const auto numLoops = sizePerBlock / blockDim.x;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        output[blockOffset + blockDim.x * i + threadIdx.x] =
            input[blockOffset + blockDim.x * i + threadIdx.x] > 0.0f ? 1.0f
                                                                     : 0.0f;
    }
}

__global__ void LeakyReLUKernel(float* output, const float* input, float a,
                                unsigned int totalSize)
{
    const auto sizePerBlock = totalSize / gridDim.x;
    const auto numLoops = sizePerBlock / blockDim.x;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        output[blockOffset + blockDim.x * i + threadIdx.x] =
            input[blockOffset + blockDim.x * i + threadIdx.x] > 0
                ? input[blockOffset + blockDim.x * i + threadIdx.x]
                : a * input[blockOffset + blockDim.x * i + threadIdx.x];
    }
}

__global__ void LeakyReLUDerivativeKernel(float* output, const float* input,
                                          const float a, unsigned int totalSize)
{
    const auto sizePerBlock = totalSize / gridDim.x;
    const auto numLoops = sizePerBlock / blockDim.x;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        output[blockOffset + blockDim.x * i + threadIdx.x] =
            input[blockOffset + blockDim.x * i + threadIdx.x] > 0.0f ? 1.0f : a;
    }
}

__global__ void InverseKernel(float* output, const float* input,
                              unsigned int totalSize)
{
    const auto sizePerBlock = totalSize / gridDim.x;
    const auto numLoops = sizePerBlock / blockDim.x;
    const auto blockOffset = sizePerBlock * blockIdx.x;

    for (unsigned int i = 0; i < numLoops; i++)
    {
        output[blockOffset + blockDim.x * i + threadIdx.x] =
            1 / input[blockOffset + blockDim.x * i + threadIdx.x];
    }
}

__global__ void MeanKernel(float* output, const float* input,
                           unsigned int totalSize, unsigned int unitSize)
{
    const auto unitId = blockIdx.x * blockDim.x + threadIdx.x;

    if (unitId < totalSize)
    {
        for (unsigned int i = 0; i < unitSize; i++)
        {
            output[unitId] += input[unitSize * unitId + i];
        }
        output[unitId] /= static_cast<float>(unitSize);
    }
}
}  // namespace Motutapu::Compute::Cuda::Dense