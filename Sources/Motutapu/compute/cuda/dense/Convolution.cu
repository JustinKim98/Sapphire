// Copyright (c) 2021, Jaewoo Kim

// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include <Motutapu/compute/cuda/dense/Convolution.cuh>

namespace Motutapu::Compute::Cuda
{
__host__ void CreateConvDescriptors(CudnnConvMetaData* metadata,
                                    Shape4D inputShape, Shape4D filterShape,
                                    int strideRow, int strideCol,
                                    int dilationRow, int dilationCol,
                                    int paddingRow, int paddingCol)
{
    int outputN = inputShape.N;
    int outputChannels = filterShape.N;
    int outputHeight = (inputShape.Height + 2 * paddingRow -
                        dilationRow * (filterShape.Height - 1) - 1) /
                           strideRow +
                       1;
    int outputWidth = (inputShape.Width + 2 * paddingCol -
                       dilationCol * (filterShape.Width - 1) - 1) /
                          strideCol +
                      1;

    checkCuDNN(hipdnnCreate(&metadata->Handle));

    checkCuDNN(hipdnnCreateConvolutionDescriptor(&metadata->ConvDesc));
    checkCuDNN(hipdnnCreateTensorDescriptor(&metadata->InputDesc));
    checkCuDNN(hipdnnCreateFilterDescriptor(&metadata->FilterDesc));
    checkCuDNN(hipdnnCreateTensorDescriptor(&metadata->OutputDesc));

    checkCuDNN(hipdnnSetConvolution2dDescriptor(
        metadata->ConvDesc, paddingRow, paddingCol, strideRow, strideCol,
        dilationRow, dilationCol, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    checkCuDNN(hipdnnSetTensor4dDescriptor(
        metadata->InputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, inputShape.N,
        inputShape.Channels, inputShape.Height, inputShape.Width));

    checkCuDNN(hipdnnSetFilter4dDescriptor(
        metadata->FilterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
        filterShape.N, filterShape.Channels, filterShape.Height,
        filterShape.Width));

    checkCuDNN(hipdnnSetTensor4dDescriptor(
        metadata->OutputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, outputN,
        outputChannels, outputHeight, outputWidth));

    int numAlgo;

    hipdnnConvolutionFwdAlgoPerf_t forwardPerf[3];
    checkCuDNN(cudnnGetConvolutionForwardAlgorithm_v7(
        metadata->Handle, metadata->InputDesc, metadata->FilterDesc,
        metadata->ConvDesc, metadata->OutputDesc, 1, &numAlgo, forwardPerf));
    metadata->ForwardAlgo = forwardPerf[0].algo;

    hipdnnConvolutionBwdDataAlgoPerf_t backDataPerf[3];
    checkCuDNN(cudnnGetConvolutionBackwardDataAlgorithm_v7(
        metadata->Handle, metadata->FilterDesc, metadata->OutputDesc,
        metadata->ConvDesc, metadata->InputDesc, 3, &numAlgo, backDataPerf));
    metadata->BackwardDataAlgo = backDataPerf[0].algo;

    hipdnnConvolutionBwdFilterAlgoPerf_t backFilterPerf[3];
    checkCuDNN(cudnnGetConvolutionBackwardFilterAlgorithm_v7(
        metadata->Handle, metadata->InputDesc, metadata->OutputDesc,
        metadata->ConvDesc, metadata->FilterDesc, 3, &numAlgo, backFilterPerf));
    metadata->BackwardFilterAlgo = backFilterPerf[0].algo;

    checkCuDNN(hipdnnGetConvolutionForwardWorkspaceSize(
        metadata->Handle, metadata->InputDesc, metadata->FilterDesc,
        metadata->ConvDesc, metadata->OutputDesc, metadata->ForwardAlgo,
        &metadata->ForwardWorkSpaceBytes));

    checkCuDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(
        metadata->Handle, metadata->FilterDesc, metadata->OutputDesc,
        metadata->ConvDesc, metadata->InputDesc, metadata->BackwardDataAlgo,
        &metadata->BackwardDataWorkSpaceBytes));

    checkCuDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
        metadata->Handle, metadata->InputDesc, metadata->OutputDesc,
        metadata->ConvDesc, metadata->FilterDesc, metadata->BackwardFilterAlgo,
        &metadata->BackwardFilterWorkSpaceBytes));

    checkCuda(hipMalloc((void**)&metadata->ForwardWorkSpace,
                         metadata->ForwardWorkSpaceBytes));

    checkCuda(hipMalloc((void**)&metadata->BackwardDataWorkSpace,
                         metadata->BackwardDataWorkSpaceBytes));

    checkCuda(hipMalloc((void**)&metadata->BackwardFilterWorkSpace,
                         metadata->BackwardFilterWorkSpaceBytes));
}

__host__ void ConvolutionForward2D(CudnnConvMetaData* metadata, float* output,
                                   float* input, float* filter,
                                   Shape4D inputShape, Shape4D filterShape,
                                   int strideRow, int strideCol,
                                   int dilationRow, int dilationCol,
                                   int paddingRow, int paddingCol)
{
    float alpha = 1.0f;
    float beta = 0.0f;
    checkCuDNN(hipdnnConvolutionForward(
        metadata->Handle, &alpha, metadata->InputDesc, input,
        metadata->FilterDesc, filter, metadata->ConvDesc, metadata->ForwardAlgo,
        metadata->ForwardWorkSpace, metadata->ForwardWorkSpaceBytes, &beta,
        metadata->OutputDesc, output));
}

__host__ void ConvolutionBackward2D(
    CudnnConvMetaData* descriptors, float* dataGradientOut, float* filter,
    float* filterGradientOut, float* input, float* gradientInput,
    Shape4D inputShape, Shape4D filterShape, int strideRow, int strideCol,
    int dilationRow, int dilationCol, int paddingRow, int paddingCol)
{
    float alpha = 1.0f;
    float beta = 0.0f;
    checkCuDNN(hipdnnConvolutionBackwardData(
        descriptors->Handle, &alpha, descriptors->FilterDesc, filter,
        descriptors->OutputDesc, gradientInput, descriptors->ConvDesc,
        descriptors->BackwardDataAlgo, descriptors->BackwardDataWorkSpace,
        descriptors->BackwardDataWorkSpaceBytes, &beta, descriptors->InputDesc,
        dataGradientOut));

    checkCuDNN(hipdnnConvolutionBackwardFilter(
        descriptors->Handle, &alpha, descriptors->InputDesc, input,
        descriptors->OutputDesc, gradientInput, descriptors->ConvDesc,
        descriptors->BackwardFilterAlgo, descriptors->BackwardFilterWorkSpace,
        descriptors->BackwardFilterWorkSpaceBytes, &beta,
        descriptors->FilterDesc, filterGradientOut));
}
}  // namespace Motutapu::Compute::Cuda