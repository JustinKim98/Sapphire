#include "hip/hip_runtime.h"
// Copyright (c) 2021, Justin Kim

// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include <hipblas.h>
#include <hipDNN.h>
#include <Motutapu/compute/cuda/Memory.cuh>
#include <Motutapu/compute/cuda/dense/Basic.cuh>
#include <Motutapu/compute/cuda/dense/BasicKernel.cuh>

namespace Motutapu::Compute::Cuda::Dense
{
__host__ void Add(unsigned int totalSize, float* output, const float* inputA,
                  const float* inputB, unsigned int inputStride,
                  bool broadcastInputA, bool broadcastInputB)
{
    //    if (broadcastInputA || broadcastInputB)
    //    {
    //        const auto firstLaunchSizeBroadcast = (inputStride / 256 + 1);
    //
    //        AddKernelBroadcast<<<(inputStride / 256 + 1), 256,
    //                             2 * 256 * sizeof(float)>>>(
    //            output, inputA, inputB, 0, totalSize, inputStride,
    //            broadcastInputA, broadcastInputB);
    //    }
    //    else
    //    {
    //        const auto numLoops = 4;
    //        const auto blockDim = 256 / numLoops;
    //
    //        auto gridDim = totalSize / 256;
    //        const auto firstLaunchSize = (gridDim + 1) * 256;
    //
    //        if (firstLaunchSize > 0)
    //            AddKernelShared<<<gridDim + 1, blockDim, 2 * 256 *
    //            sizeof(float)>>>(
    //                output, inputA, inputB, 0, firstLaunchSize, totalSize,
    //                inputStride, numLoops, broadcastInputA, broadcastInputB);
    //
    //        hipDeviceSynchronize();
    //    }

    const auto numLoops = 8;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto blockDim = totalSize / MAX_THREAD_DIM_X;
    const auto firstLaunchSize = blockDim * MAX_THREAD_DIM_X;

    if (firstLaunchSize > 0)
        AddKernel<<<blockDim, threadDim>>>(
            output, inputA, inputB, 0, firstLaunchSize, totalSize, inputStride,
            broadcastInputA, broadcastInputB);

    if (totalSize > firstLaunchSize)
    {
        const unsigned int offset = firstLaunchSize;

        AddKernel<<<1, totalSize - firstLaunchSize>>>(
            output, inputA, inputB, offset, totalSize - firstLaunchSize,
            totalSize, inputStride, broadcastInputA, broadcastInputB);
    }
}

__host__ void Sub(unsigned int totalSize, float* output, const float* inputA,
                  const float* inputB, unsigned int inputStride,
                  bool broadcastInputA, bool broadcastInputB)
{
    const auto numLoops = 8;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto blockDim = totalSize / MAX_THREAD_DIM_X;
    const auto firstLaunchSize = blockDim * MAX_THREAD_DIM_X;

    if (firstLaunchSize > 0)
        SubKernel<<<blockDim, threadDim>>>(
            output, inputA, inputB, 0, firstLaunchSize, totalSize, inputStride,
            broadcastInputA, broadcastInputB);

    if (totalSize > firstLaunchSize)
    {
        const unsigned int offset = firstLaunchSize;

        SubKernel<<<1, totalSize - firstLaunchSize>>>(
            output, inputA, inputB, offset, totalSize - firstLaunchSize,
            totalSize, inputStride, broadcastInputA, broadcastInputB);
    }
}

__host__ void Scale(float* output, const float* input, const float scaleFactor,
                    unsigned int totalSize)
{
    const auto numLoops = 8;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto blockDim = totalSize / (threadDim * numLoops);
    const auto firstLaunchSize = blockDim * threadDim * numLoops;

    if (firstLaunchSize > 0)
        ScaleKernel<<<blockDim, threadDim>>>(output, input, scaleFactor,
                                             firstLaunchSize);
    if (totalSize > firstLaunchSize)
    {
        const float* inputOffset = input + firstLaunchSize;
        float* outputOffset = output + firstLaunchSize;

        ScaleKernel<<<1, totalSize - firstLaunchSize>>>(
            outputOffset, inputOffset, scaleFactor,
            totalSize - firstLaunchSize);
    }
}

__host__ void Transpose(float* output, const float* input,
                        unsigned int inputNumRows, unsigned int inputNumCols,
                        unsigned int batchSize, bool broadcastInput)
{
    unsigned int blockDimX = (inputNumCols % TILE_DIM == 0)
                                 ? inputNumCols / TILE_DIM
                                 : inputNumCols / TILE_DIM + 1;
    unsigned int blockDimY = (inputNumRows % TILE_DIM == 0)
                                 ? inputNumRows / TILE_DIM
                                 : inputNumRows / TILE_DIM + 1;

    unsigned int blockDimZ = batchSize;
    dim3 blockDim(blockDimX, blockDimY, blockDimZ);
    dim3 threadDim(TILE_DIM, 8);
    TransposeKernel<<<blockDim, threadDim>>>(output, input, inputNumRows,
                                             inputNumCols, broadcastInput);
}

__host__ void Dot(unsigned int totalSize, float* output, const float* inputA,
                  const float* inputB, unsigned int inputStride,
                  bool broadcastInputA, bool broadcastInputB)
{
    const auto numLoops = 8;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto blockDim = totalSize / MAX_THREAD_DIM_X;
    const auto firstLaunchSize = blockDim * MAX_THREAD_DIM_X;

    if (firstLaunchSize > 0)
        DotKernel<<<blockDim, threadDim>>>(
            output, inputA, inputB, 0, firstLaunchSize, totalSize, inputStride,
            broadcastInputA, broadcastInputB);

    if (totalSize > firstLaunchSize)
    {
        const unsigned int offset = firstLaunchSize;

        DotKernel<<<1, totalSize - firstLaunchSize>>>(
            output, inputA, inputB, offset, totalSize - firstLaunchSize,
            totalSize, inputStride, broadcastInputA, broadcastInputB);
    }
}

__host__ void Pow(float* output, const float* input, const float factor,
                  unsigned int totalSize)
{
    const auto numLoops = 8;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto blockDim = totalSize / (threadDim * numLoops);
    const auto firstLaunchSize = blockDim * threadDim * numLoops;

    if (firstLaunchSize > 0)
        PowKernel<<<blockDim, threadDim>>>(output, input, factor,
                                           firstLaunchSize);
    if (totalSize > firstLaunchSize)
    {
        const float* inputOffset = input + firstLaunchSize;
        float* outputOffset = output + firstLaunchSize;

        PowKernel<<<1, totalSize - firstLaunchSize>>>(
            outputOffset, inputOffset, factor, totalSize - firstLaunchSize);
    }
}

__host__ void cos(float* output, const float* input, unsigned int totalSize)
{
    const auto numLoops = 8;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto blockDim = totalSize / (threadDim * numLoops);
    const auto firstLaunchSize = blockDim * threadDim * numLoops;

    if (firstLaunchSize > 0)
        cosKernel<<<blockDim, threadDim>>>(output, input, firstLaunchSize);
    if (totalSize > firstLaunchSize)
    {
        const float* inputOffset = input + firstLaunchSize;
        float* outputOffset = output + firstLaunchSize;

        cosKernel<<<1, totalSize - firstLaunchSize>>>(
            outputOffset, inputOffset, totalSize - firstLaunchSize);
    }
}

__host__ void sin(float* output, const float* input, unsigned int totalSize)
{
    const auto numLoops = 8;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto blockDim = totalSize / (threadDim * numLoops);
    const auto firstLaunchSize = blockDim * threadDim * numLoops;

    if (firstLaunchSize > 0)
        sinKernel<<<blockDim, threadDim>>>(output, input, firstLaunchSize);
    if (totalSize > firstLaunchSize)
    {
        const float* inputOffset = input + firstLaunchSize;
        float* outputOffset = output + firstLaunchSize;

        sinKernel<<<1, totalSize - firstLaunchSize>>>(
            outputOffset, inputOffset, totalSize - firstLaunchSize);
    }
}

__host__ void tan(float* output, const float* input, unsigned int totalSize)
{
    const auto numLoops = 8;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto blockDim = totalSize / (threadDim * numLoops);
    const auto firstLaunchSize = blockDim * threadDim * numLoops;

    if (firstLaunchSize > 0)
        tanKernel<<<blockDim, threadDim>>>(output, input, firstLaunchSize);
    if (totalSize > firstLaunchSize)
    {
        const float* inputOffset = input + firstLaunchSize;
        float* outputOffset = output + firstLaunchSize;

        tanKernel<<<1, totalSize - firstLaunchSize>>>(
            outputOffset, inputOffset, totalSize - firstLaunchSize);
    }
}

__host__ void cosh(float* output, const float* input, unsigned int totalSize)
{
    const auto numLoops = 8;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto blockDim = totalSize / (threadDim * numLoops);
    const auto firstLaunchSize = blockDim * threadDim * numLoops;

    if (firstLaunchSize > 0)
        coshKernel<<<blockDim, threadDim>>>(output, input, firstLaunchSize);
    if (totalSize > firstLaunchSize)
    {
        const float* inputOffset = input + firstLaunchSize;
        float* outputOffset = output + firstLaunchSize;

        cosKernel<<<1, totalSize - firstLaunchSize>>>(
            outputOffset, inputOffset, totalSize - firstLaunchSize);
    }
}

__host__ void sinh(float* output, const float* input, unsigned int totalSize)
{
    const auto numLoops = 8;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto blockDim = totalSize / (threadDim * numLoops);
    const auto firstLaunchSize = blockDim * threadDim * numLoops;

    if (firstLaunchSize > 0)
        sinhKernel<<<blockDim, threadDim>>>(output, input, firstLaunchSize);
    if (totalSize > firstLaunchSize)
    {
        const float* inputOffset = input + firstLaunchSize;
        float* outputOffset = output + firstLaunchSize;

        sinhKernel<<<1, totalSize - firstLaunchSize>>>(
            outputOffset, inputOffset, totalSize - firstLaunchSize);
    }
}

__host__ void tanh(float* output, const float* input, unsigned int totalSize)
{
    const auto numLoops = 8;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto blockDim = totalSize / (threadDim * numLoops);
    const auto firstLaunchSize = blockDim * threadDim * numLoops;

    if (firstLaunchSize > 0)
        tanhKernel<<<blockDim, threadDim>>>(output, input, firstLaunchSize);
    if (totalSize > firstLaunchSize)
    {
        const float* inputOffset = input + firstLaunchSize;
        float* outputOffset = output + firstLaunchSize;

        tanhKernel<<<1, totalSize - firstLaunchSize>>>(
            outputOffset, inputOffset, totalSize - firstLaunchSize);
    }
}

__host__ void log(float* output, const float* input, unsigned int totalSize)
{
    const auto numLoops = 8;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto blockDim = totalSize / (threadDim * numLoops);
    const auto firstLaunchSize = blockDim * threadDim * numLoops;

    if (firstLaunchSize > 0)
        logKernel<<<blockDim, threadDim>>>(output, input, firstLaunchSize);
    if (totalSize > firstLaunchSize)
    {
        const float* inputOffset = input + firstLaunchSize;
        float* outputOffset = output + firstLaunchSize;

        logKernel<<<1, totalSize - firstLaunchSize>>>(
            outputOffset, inputOffset, totalSize - firstLaunchSize);
    }
}

__host__ void log10(float* output, const float* input, unsigned int totalSize)
{
    const auto numLoops = 8;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto blockDim = totalSize / (threadDim * numLoops);
    const auto firstLaunchSize = blockDim * threadDim * numLoops;

    if (firstLaunchSize > 0)
        log10Kernel<<<blockDim, threadDim>>>(output, input, firstLaunchSize);
    if (totalSize > firstLaunchSize)
    {
        const float* inputOffset = input + firstLaunchSize;
        float* outputOffset = output + firstLaunchSize;

        log10Kernel<<<1, totalSize - firstLaunchSize>>>(
            outputOffset, inputOffset, totalSize - firstLaunchSize);
    }
}

__host__ void ReLU(float* output, const float* input, unsigned int totalSize)
{
    const auto numLoops = 8;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto blockDim = totalSize / (threadDim * numLoops);
    const auto firstLaunchSize = blockDim * threadDim * numLoops;

    if (firstLaunchSize > 0)
        ReLUKernel<<<blockDim, threadDim>>>(output, input, firstLaunchSize);
    if (totalSize > firstLaunchSize)
    {
        const float* inputOffset = input + firstLaunchSize;
        float* outputOffset = output + firstLaunchSize;

        ReLUKernel<<<1, totalSize - firstLaunchSize>>>(
            outputOffset, inputOffset, totalSize - firstLaunchSize);
    }
}

__host__ void ReLUDerivative(float* output, const float* input,
                             unsigned int totalSize)
{
    const auto numLoops = 8;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto blockDim = totalSize / (threadDim * numLoops);
    const auto firstLaunchSize = blockDim * threadDim * numLoops;

    if (firstLaunchSize > 0)
        ReLUDerivativeKernel<<<blockDim, threadDim>>>(output, input,
                                                      firstLaunchSize);
    if (totalSize > firstLaunchSize)
    {
        const float* inputOffset = input + firstLaunchSize;
        float* outputOffset = output + firstLaunchSize;

        ReLUDerivativeKernel<<<1, totalSize - firstLaunchSize>>>(
            outputOffset, inputOffset, totalSize - firstLaunchSize);
    }
}

__host__ void LeakyReLU(float* output, const float* input, const float a,
                        unsigned int totalSize)
{
    const auto numLoops = 8;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto blockDim = totalSize / (threadDim * numLoops);
    const auto firstLaunchSize = blockDim * threadDim * numLoops;

    if (firstLaunchSize > 0)
        LeakyReLUKernel<<<blockDim, threadDim>>>(output, input, a,
                                                 firstLaunchSize);
    if (totalSize > firstLaunchSize)
    {
        const float* inputOffset = input + firstLaunchSize;
        float* outputOffset = output + firstLaunchSize;

        LeakyReLUKernel<<<1, totalSize - firstLaunchSize>>>(
            outputOffset, inputOffset, a, totalSize - firstLaunchSize);
    }
}

__host__ void LeakyReLUDerivative(float* output, const float* input,
                                  const float a, unsigned int totalSize)
{
    const auto numLoops = 8;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto blockDim = totalSize / (threadDim * numLoops);
    const auto firstLaunchSize = blockDim * threadDim * numLoops;

    if (firstLaunchSize > 0)
        LeakyReLUDerivativeKernel<<<blockDim, threadDim>>>(output, input, a,
                                                           firstLaunchSize);
    if (totalSize > firstLaunchSize)
    {
        const float* inputOffset = input + firstLaunchSize;
        float* outputOffset = output + firstLaunchSize;

        LeakyReLUDerivativeKernel<<<1, totalSize - firstLaunchSize>>>(
            outputOffset, inputOffset, a, totalSize - firstLaunchSize);
    }
}

__host__ void Inverse(float* output, const float* input, unsigned int totalSize)
{
    const auto numLoops = 8;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto blockDim = totalSize / (threadDim * numLoops);
    const auto firstLaunchSize = blockDim * threadDim * numLoops;

    if (firstLaunchSize > 0)
        InverseKernel<<<blockDim, threadDim>>>(output, input, firstLaunchSize);
    if (totalSize > firstLaunchSize)
    {
        const float* inputOffset = input + firstLaunchSize;
        float* outputOffset = output + firstLaunchSize;

        InverseKernel<<<1, totalSize - firstLaunchSize>>>(
            outputOffset, inputOffset, totalSize - firstLaunchSize);
    }
}

//! output size should be totalSize/unitSize
__host__ void Mean(float* output, const float* input, unsigned int totalSize,
                   unsigned int unitSize)
{
    const auto numLoops = 8;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto requiredThreadNum = totalSize / unitSize;
    const auto blockDim = requiredThreadNum / (threadDim * numLoops);
    const auto firstLaunchSize = blockDim * threadDim * numLoops;

    if (firstLaunchSize > 0)
        MeanKernel<<<blockDim, threadDim>>>(output, input, firstLaunchSize);
    if (requiredThreadNum > firstLaunchSize)
    {
        const float* inputOffset = input + firstLaunchSize;
        float* outputOffset = output + firstLaunchSize;

        MeanKernel<<<1, requiredThreadNum - firstLaunchSize>>>(
            output, input, totalSize, unitSize);
    }
}

//__global__ void ConvInputToFeatureMatrix(
//    float* out, float* input, unsigned int inputChannels,
//    unsigned int inputRows, unsigned int inputColumns,
//    unsigned int inputPaddedRows, unsigned int inputPaddedColumns,
//    unsigned int outputPaddedRows, unsigned int outputPaddedColumns,
//    unsigned int filterRows, unsigned int filterCols, unsigned int padSizeRow,
//    unsigned int padSizeCol, unsigned int strideRow, unsigned int strideCol,
//    unsigned int dilationRow, unsigned int dilationCol)
//{
//    const int threadIdx = blockIdx.x * blockDim.x + threadIdx.x;
//
//    const int inputMatrixSize = inputPaddedRows * inputPaddedColumns;
//    const int convPerRow =
//        (inputRows - filterRows + 1 + padSizeRow * 2) / strideRow;
//    const int convPerCol =
//        (inputCols - filterCols + 1 + padSizeCol * 2) / strideCol;
//
//    const int channelIdx = threadIdx / convPerRow * convPerCol;
//    const int convRowIdx = (threadIdx % convPerRow * convPerCol) / convPerRow;
//    const int convColIdx = (threadIdx % convPerRow * convPerCol) / convPerCol;
//
//    float* inputStartOffset = input + inputMatrixSize * channelIdx +
//                              inputPaddedColumns * strideRow * convRowIdx +
//                              strideCol * convColIdx;
//
//    float* outputStartOffset =
//        output + outputPaddedColumns * (convPerRow * convRowIdx + convColIdx)
//        + filterRows * filterCols * channelIdx;
//
//    for (int i = 0; i < filterRows; i++)
//        for (j = 0; j < filterCols; j++)
//        {
//            *(inputStartOffset + inputPaddedColumns * i + j)
//        }
//}
}  // namespace Motutapu::Compute::Cuda::Dense