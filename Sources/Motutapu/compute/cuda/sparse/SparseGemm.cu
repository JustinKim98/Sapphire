#include "hip/hip_runtime.h"
// Copyright (c) 2021, Justin Kim

// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include <stdint-gcc.h>
#include <Motutapu/compute/cuda/sparse/SparseGemm.cuh>
#include <cstdlib>

#define MAX_NNZ_PER_BLOCK_LARGE 1024
#define MAX_NNZ_PER_BLOCK_SMALL 512
#define GEMM_BLOCK_NUM 16
#define MAX_BLOCK_DIM 1024

namespace Motutapu::Compute::Sparse
{
__host__ void Gemm(SparseMatrix* output, SparseMatrix* a, SparseMatrix* b,
                   LoadDistMatrix* loadDist, size_t numMatrices)
{
    auto* nnzArray =
        static_cast<uint32_t*>(malloc(sizeof(uint32_t) * numMatrices));
    CallLoadDist(a, b, loadDist, nnzArray, numMatrices);
    AllocateOutput(output, a, b, numMatrices, nnzArray);
}

__host__ void CallLoadDist(SparseMatrix* a, SparseMatrix* b,
                           LoadDistMatrix* loadDist, uint32_t* nnzArray,
                           size_t numMatrices)
{
    const auto numLoops = 8;
    const auto M = a[0].M;
    const auto threadDim = M / numLoops;
    uint32_t* deviceNNZArray = nullptr;
    hipMalloc((void**)&deviceNNZArray, sizeof(uint32_t) * numMatrices);

    const auto blockDim =
        (numMatrices > MAX_GRID_DIM) ? numMatrices - MAX_GRID_DIM : numMatrices;

    if (blockDim > 0)
        LoadDistKernel<<<blockDim, threadDim>>>(loadDist, a, b, deviceNNZArray);
    if (numMatrices > blockDim)
    {
        const SparseMatrix* offsetA = a + blockDim;
        const SparseMatrix* offsetB = b + blockDim;
        LoadDistMatrix* loadDistOffset = loadDist + blockDim;

        const auto secondBlockDim = numMatrices - blockDim;
        LoadDistKernel<<<secondBlockDim, threadDim>>>(loadDistOffset, offsetA,
                                                      offsetB, deviceNNZArray);
    }
    hipMemcpy(nnzArray, deviceNNZArray, sizeof(uint32_t) * numMatrices,
               hipMemcpyDeviceToHost);
    hipFree(deviceNNZArray);
}

__host__ void AllocateOutput(SparseMatrix* output, SparseMatrix* a,
                             SparseMatrix* b, size_t numMatrices,
                             const uint32_t* nnzArray)
{
    for (uint32_t matrixIdx = 0; matrixIdx < numMatrices; ++matrixIdx)
    {
        SparseMatrix* curOutput = output + matrixIdx;
        curOutput->M = a->M;
        curOutput->N = b->N;
        curOutput->NNZ = nnzArray[matrixIdx];
        hipFree(curOutput->V);
        hipFree(curOutput->ROW);
        hipFree(curOutput->COL);
        hipMalloc((void**)curOutput->V, sizeof(float) * curOutput->NNZ);
        hipMalloc((void**)curOutput->COL, sizeof(float) * curOutput->NNZ);
        hipMalloc((void**)curOutput->ROW,
                   sizeof(uint32_t) * (curOutput->M + 1));
    }
}

//! Todo : unify calculate Load kernel and Calculate Gemm
//! Should be executed using single block
__global__ void LoadDistKernel(LoadDistMatrix* loadDist, SparseMatrix* a,
                               SparseMatrix* b, uint32_t* nnzArray)
{
    __shared__ uint32_t* nnzPerMatrix;

    uint32_t rowStart[GEMM_BLOCK_NUM];
    uint32_t colStart[GEMM_BLOCK_NUM];
    //! ByteSize must be larger than Number of required blocks per row + 1

    const auto matrixIdx = blockIdx.x;
    const auto rowIdxBegin = threadIdx.x;
    const auto rowIdxStride = blockDim.x;

    SparseMatrix* curA = a + matrixIdx;
    SparseMatrix* curB = b + matrixIdx;
    LoadDistMatrix* curLoadDist = loadDist + matrixIdx;

    if (threadIdx.x == 0)
        *nnzPerMatrix = 0;

    uint32_t idx = 0;
    for (auto rowIdxA = rowIdxBegin; rowIdxA < a[matrixIdx].M;
         rowIdxA += rowIdxStride)
    {
        auto sparseColIdxA = curA->ROW[rowIdxA];
        curLoadDist->ROW[rowIdxA] = curA->ROW[rowIdxA];
        rowStart[idx] = rowIdxA;
        colStart[idx] = sparseColIdxA;
        uint32_t curLoad = 0;
        uint32_t nnzPerRow = 0;
        for (; sparseColIdxA < curA->ROW[rowIdxA + 1]; ++sparseColIdxA)
        {
            const auto colIdxA = curA->COL[sparseColIdxA];
            const auto numElemPerRowB =
                curB->ROW[colIdxA + 1] - curB->ROW[colIdxA];
            curLoadDist->Load[sparseColIdxA] = numElemPerRowB;
            curLoadDist->COL[sparseColIdxA] = colIdxA;

            if (sparseColIdxA != a->ROW[rowIdxA])
            {
                //! Load will stack as row advances
                curLoadDist->Load[sparseColIdxA] +=
                    curLoadDist->Load[sparseColIdxA - 1];
            }

            if (curLoad + numElemPerRowB > MAX_NNZ_PER_BLOCK_LARGE)
            {
                idx += 1;
                rowStart[idx] = rowIdxA;
                colStart[idx] = sparseColIdxA;
                curLoad = 0;
            }
            curLoad += numElemPerRowB;
            nnzPerRow += numElemPerRowB;
        }
        atomicAdd_block(nnzPerMatrix, nnzPerRow);
    }

    curLoadDist->NNZ = *nnzPerMatrix;
    nnzArray[matrixIdx] = *nnzPerMatrix;
}

__global__ void CalculateRowKernel(SparseMatrix* out, SparseMatrix* a,
                                   SparseMatrix* b, LoadDistMatrix* loadDist,
                                   uint32_t rowIdx, uint32_t sparseColIdxBegin,
                                   uint32_t sparseColIdxEnd)
{
    //! Stores pair of computer value and pair of index
    __shared__ float tempValueArray[MAX_NNZ_PER_BLOCK_LARGE];
    __shared__ uint32_t tempIdxArray[MAX_NNZ_PER_BLOCK_LARGE];

    const auto M = out[0].M;
    SparseMatrix* curOut = out + blockIdx.x / M;
    SparseMatrix* curA = a + blockIdx.x / M;
    SparseMatrix* curB = b + blockIdx.x / M;
    LoadDistMatrix* curLoadDist = loadDist + blockIdx.x / M;

    const auto sparseColIdxOffset = threadIdx.x;
    const auto sparseColIdxA = sparseColIdxBegin + sparseColIdxOffset;

    const auto colIdxA = curA->COL[sparseColIdxA];
    const auto sparseColIdxBBegin = curB->ROW[colIdxA];
    const auto sparseColIdxBEnd = curB->ROW[colIdxA + 1];
    const auto nnz = curLoadDist->Load[sparseColIdxEnd - 1];

    if (sparseColIdxA < sparseColIdxEnd)
    {
        for (uint32_t sparseColIdxB = sparseColIdxBBegin;
             sparseColIdxB < sparseColIdxBEnd; ++sparseColIdxB)
        {
            const auto unmergedSparseRowIdx = loadDist->Load[sparseColIdxA] +
                                              sparseColIdxB -
                                              sparseColIdxBBegin;

            tempValueArray[unmergedSparseRowIdx] =
                curA->V[sparseColIdxA] * curB->V[sparseColIdxB];
            tempIdxArray[unmergedSparseRowIdx] = curB->COL[sparseColIdxB];
        }
    }

    uint32_t mergedNumElements = 0;
    Sort(tempValueArray, tempIdxArray, nnz);
    Merge(tempValueArray, tempIdxArray, nnz, &mergedNumElements);

    const auto stride = (mergedNumElements / blockDim.x > 0)
                            ? mergedNumElements / blockDim.x
                            : 1;

    for (uint32_t idx = stride * threadIdx.x;
         idx < nnz && idx < stride * (threadIdx.x + 1); ++idx)
    {
        //! TODO : How are we going to store the result if row is distributed?
        const auto tempIdx = MAX_NNZ_PER_BLOCK_LARGE - nnz + idx;
        curOut->V[idx] = tempValueArray[tempIdx];
        curOut->COL[idx] = tempIdxArray[tempIdx];
    }
}

// todo : Check if this algorithm works by manipulating on python
__device__ void Sort(float* tempValArray, uint32_t* tempIdxArray,
                     uint32_t arraySize)
{
    const auto id = threadIdx.x;

    // todo : initialize array with largest negative number

    if (id > arraySize / 2)
        return;

    for (uint32_t level = 0; level < log2(arraySize); ++level)
    {
        for (auto stride = __double2uint_rz(pow(2, level)); stride > 0;
             stride /= 2)
        {
            bool dir =
                (id / stride) % 2 == 0;  // true if downward, false if upward

            if (dir)
            {
                const auto idx =
                    (id / (stride * 2)) * stride * 4 + id % (stride * 2);
                float tempVal = tempValArray[idx];
                uint32_t tempIdx = tempIdxArray[idx];

                tempValArray[idx] = tempValArray[idx + stride];
                tempIdxArray[idx] = tempIdxArray[idx + stride];

                tempValArray[idx + stride] = tempVal;
                tempIdxArray[idx + stride] = tempIdx;
            }
            else
            {
                const auto idx = (id / (stride * 2)) * stride * 4 +
                                 (stride * 4 - id % (stride * 2));
                float tempVal = tempValArray[idx];
                uint32_t tempIdx = tempIdxArray[idx];

                tempValArray[idx] = tempValArray[idx - stride];
                tempIdxArray[idx] = tempIdxArray[idx - stride];

                tempValArray[idx - stride] = tempVal;
                tempIdxArray[idx - stride] = tempIdx;
            }
        }
    }
}
}  // namespace Motutapu::Compute::Sparse