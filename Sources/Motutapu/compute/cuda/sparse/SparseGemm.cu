#include "hip/hip_runtime.h"
// Copyright (c) 2021, Justin Kim

// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include <stdint-gcc.h>
#include <Motutapu/compute/cuda/sparse/SparseGemm.cuh>

#define MAX_NNZ_PER_BLOCK_LARGE 1024
#define MAX_NNZ_PER_BLOCK_SMALL 512
#define GEMM_BLOCK_NUM 16

namespace Motutapu::Compute
{
__host__ void CalculateLoad(SparseMatrix* a, SparseMatrix* b,
                            LoadDistMatrix* loadDist, size_t numMatrices)
{
    const auto numLoops = 8;
    const auto threadDim = MAX_THREAD_DIM_X / numLoops;

    const auto blockDim = totalSize / MAX_THREAD_DIM_X;
    const auto firstLaunchSize = blockDim * threadDim * numLoops;

    if (firstLaunchSize > 0)
        CalculateLoadKernel<<<blockDim, threadDim>>>(loadDist, a, b);
    if (numMatrices > firstLaunchSize)
    {
        const SparseMatrix* offsetA = a + firstLaunchSize;
        const SparseMatrix* offsetB = b + firstLaunchSize;
        SparseMatrix* loadDistOffset = loadDist + firstLaunchSize;

        const auto secondLaunchSize = numMatrices - firstLaunchSize;

        CalculateLoadKernel<<<secondLaunchSize, threadDim>>>(loadDistOffset,
                                                             offsetA, offsetB);
    }
}

//! Todo : unify calculate Load kernel and Calculate Gemm
//! Should be executed using single block
__global__ void CalculateLoadKernel(LoadDistMatrix* loadDist, SparseMatrix* a,
                                    SparseMatrix* b)
{
    __shared__ uint32_t blockStartAddr[GEMM_BLOCK_NUM];
    //! Stores data in the format of (ROW, COL)
    //! Size must be larger than 2*(Number of required blocks + 1)

    uint32_t idx = 0;
    for (auto rowIdx = threadIdx.x; rowIdx < M; rowIdx += blockDim.x)
    {
        loadDist->ROW[rowIdx] = a->ROW[rowIdx];
        uint32_t rowLoad = 0;
        for (auto sparseColIdx = a->ROW[rowIdx];
             sparseColIdx < a->ROW[rowIdx + 1]; ++sparseColIdx)
        {
            const auto colIdx = a->COL[sparseColIdx];
            const auto numElemPerRowB = b->Row[colIdx + 1] - b->ROW[colIdx];
            loadDist->Load[sparseColIdx] = numElemPerRowB;
            loadDist->COL[sparseColIdx] = colIdx;

            if (sparseColIdx != a->ROW[rowIdx])
            {
                //! Load will stack as row advances
                loadDist->Load[sparseColIdx] +=
                    loadDist->Load[sparseColIdx - 1];
            }

            if (rowLoad + numElemPerRowB > MAX_NNZ_PER_BLOCK_LARGE)
            {
                blockStartAddr[idx] = rowIdx;
                blockStartAddr[idx + 1] = colIdx;
                idx += 2;
                rowLoad = 0;
            }
            rowLoad += numElemPerRowB;
        }
    }

    __syncthreads();

    if (threadIdx.x < idx / 2)
    {
        //! todo : Launch kernel Per block
        //! todo : How will we allocate the memory for output
        //! when we don't know how large it will be?
        //! Allocate the output with NNZ per ROW
        CalculateRowKernel<<<1, 32>>>(
            nullptr, nullptr, a, b, loadDist, blockStartAddr[threadIdx.x],
            blockStartAddr[threadIdx.x + 1], blockStartAddr[threadIdx.x + 2]);
    }
}

__global__ void CalculateRowKernel(float* cV, uint32_t* cCOL, SparseMatrix* a,
                                   SparseMatrix* b,
                                   LoadDistMatrix* stackedLoadDist,
                                   uint32_t rowIdx,
                                   uint32_t sparseColIndexBegin,
                                   uint32_t sparseColIndexEnd)
{
    //! Stores pair of computer value and pair of index
    __shared__ float tempValueArray[MAX_NNZ_PER_BLOCK_LARGE];
    __shared__ uint32_t tempIdxArray[MAX_NNZ_PER_BLOCK_LARGE];

    const auto sparseColIdxOffset = threadIdx.x;
    const auto sparseColIdxA = sparseColIdxBegin + sparseColIdxOffset;

    const auto colIdxA = a->COL[sparseColIdxA];
    const auto sparseColIdxBBegin = b->ROW[colIdxA];
    const auto sparseColIdxBEnd = b->ROW[colIdxA + 1];
    const auto nnz = stackedLoadDist->Load->V[sparseColIndexEnd - 1];

    if (sparseColIdxA < sparseColIdxAEnd)
    {
        for (uint32_t sparseColIdxB = sparseColIdxBBegin;
             sparseColIdxB < sparseColIdxBEnd, ++sparseColIdxB)
        {
            const auto unmergedSparseRowIdx =
                stackedLoadDist->Load[sparseColIdxA] + sparseColIdxB -
                sparseColIdxBBegin;

            tempValueArray[unmergedSparseRowIdx] =
                a->V[sparseColIdxA] * b->V[sparseColIdxB];
            tempIdxArray[unmergedSparseRowIdx] = b->COL[sparseColIdxB];
        }
    }

    Sort(tempValueArray, tempIdxArray, nnz);
    Merge(tempValueArray, tempIdxArray, nnz);

    const auto NNZPerThread = (nnz / threadDim.x > 0) ? nnz / threadDim.x : 1;

    for (uint32_t idx = NNZPerThread * threadIdx.x;
         idx < nnz && idx < NNZPerThread * (threadIdx.x + 1); ++idx)
    {
        const auto tempIdx = MAX_NNZ_PER_BLOCK_LARGE - nnz + idx;
        cV[idx] = tempValueArray[tempIdx];
        cCOL[idx] = tempValueArray[tempIdx];
    }
}

__host__ void CalculateGemm(SparseMatrix* c, const SparseMatrix* a,
                            const SparseMatrix* b, LoadDistMatrix* loadDist,
                            uint32_t matrixNum)
{
    for (uint32_t matrixIdx = 0; matrixIdx < matrixNum; ++matrixIdx)
    {
        for (uint32_t rowIdx = 0; rowIdx < loadDist[matrixIdx].M + 1; ++rowIdx)
        {
            uint32_t nnz = 0;
            uint32_t prevSparseColIdx = 0;
            uint32_t sparseColIdx = uint32_t sparseColIdx =
                loadDist[matrixIdx].ROW[rowIdx];
            for (; sparseColIdx < loadDist[matrixIdx].ROW[rowIdx + 1];
                 ++sparseColIdx)
            {
                if (nnz + loadDist[matrixIdx].Load[sparseColIdx] >=
                    MAX_NNZ_PER_BLOCK_LARGE)
                {
                    CalculateRowKernel<<<1, requiredThreads>>>(
                        c + matrixIdx, nullptr, 0, a + matrixIdx, b + matrixIdx,
                        loadDist + matrixIdx, rowIdx, prevSparseColIdx);
                    prevSparseColIdx = sparseColIdx;
                    nnz = 0;
                }

                nnz += loadDist[matrixIdx].Load[sparseColIdx];
                loadDist[matrixIdx].Load[sparseColIdx] = nnz;
            }

            if (nnz > 0 && nnz <= MAX_NNZ_PER_BLOCK_SMALL)
            {
            }
            else if (nnz > MAX_NNZ_PER_BLOCK_SMALL)
            {
                CalculateRowKernel<<<1, requiredThreads>>>(
                    c + matrixIdx, nullptr, 0, a + matrixIdx, b + matrixIx,
                    loadDist + matrixIdx, rowIdx, prevSparseColIdx);
                prevSparseColIdx = sparseColIdx;
            }
        }
    }
}

}  // namespace Motutapu::Compute