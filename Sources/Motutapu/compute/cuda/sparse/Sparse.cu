#include "hip/hip_runtime.h"
// Copyright (c) 2021, Justin Kim

// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include <Motutapu/compute/cuda/CudaParams.cuh>
#include <Motutapu/compute/cuda/sparse/Sparse.cuh>

namespace Motutapu::Compute
{
__host__ void DeepAllocateSparseMatrix(SparseMatrix* cudaTarget, size_t m,
                                       size_t nnz)
{
    hipMalloc(&cudaTarget->V, sizeof(float) * nnz);
    hipMalloc(&cudaTarget->COL, sizeof(uint32_t) * nnz);
    hipMalloc(&cudaTarget->ROW, sizeof(uint32_t) * m);
    cudaTarget->M = m;
    cudaTarget->NNZ = nnz;
}

__host__ void ShallowAllocateSparseMatrix(SparseMatrix* target)
{
    hipMalloc(target, sizeof(SparseMatrix));
}

__host__ void DeepFreeSparseMatrix(SparseMatrix* target)
{
    hipFree(&dest->COL);
    hipFree(&dest->ROW);
    hipFree(&dest);
}

__host__ void ShallowFreeSparseMatrix(SparseMatrix* target)
{
    hipFree(&dest->V);
    hipFree(&dest->COL);
    hipFree(&dest->ROW);
}

__global__ void ConvertDenseToSparseKernel(SparseMatrix* dst, float* src,
                                           uint32_t numRows, uint32_t numCols,
                                           uint32_t numMatrices)
{

}

__global__ void ConvertSparseToDenseKernel(float* dst, SparseMatrix* src,
                                           uint32_t numMatrices)
{
}

}  // namespace Motutapu::Compute