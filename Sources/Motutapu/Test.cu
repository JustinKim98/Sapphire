
#include <iostream>
#include <Motutapu/Test.hpp>
#include <hip/hip_runtime.h>

void PrintCudaVersion()
{
    int runtime_ver;
    hipRuntimeGetVersion(&runtime_ver);
    std::cout << "CUDA Runtime version: " << runtime_ver << std::endl;

    int driver_ver;
    hipDriverGetVersion(&driver_ver);
    std::cout << "CUDA Driver version: " << driver_ver << std::endl;
}

