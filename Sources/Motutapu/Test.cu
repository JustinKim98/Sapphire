//#ifdef WITH_CUDA

#include <hip/hip_runtime.h>
#include <iostream>

namespace Test::Cuda
{
void PrintCudaVersion()
{
    int runtime_ver;
    hipRuntimeGetVersion(&runtime_ver);
    std::cout << "CUDA Runtime version: " << runtime_ver << std::endl;

    int driver_ver;
    hipDriverGetVersion(&driver_ver);
    std::cout << "CUDA Driver version: " << driver_ver << std::endl;

    float* toMalloc;
    auto error = hipMalloc((void**)&toMalloc, 100 * sizeof(float));
    if (error != hipSuccess)
        throw std::runtime_error("Allocation failure");

    error = hipFree(toMalloc);
    if (error != hipSuccess)
        throw std::runtime_error("Free failure");

    std::cout << "CudaSuccess" << std::endl;
}
}  // namespace Test::Cuda
   //#endif