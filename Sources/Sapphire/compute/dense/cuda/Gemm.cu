// Copyright (c) 2021, Justin Kim

// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include <hipblas.h>
#include <Sapphire/compute/cudaUtil/CudaParams.cuh>
#include <Sapphire/compute/cudaUtil/Memory.hpp>
#include <Sapphire/util/ResourceManager.hpp>
#include <Sapphire/compute/dense/cuda/Gemm.cuh>
#include <Sapphire/compute/dense/cuda/kernels/GemmKernel.cuh>
#include <cassert>

namespace Sapphire::Compute::Dense::Cuda
{
//! All size parameters should be at least 1
//! batch sizes must be multiple of each other
__host__ void Gemm(unsigned int totalSize, float* out, const float* A,
                   const float* B,
                   const float* C, unsigned int M, unsigned int N,
                   unsigned int K,
                   int deviceId)
{
    const auto tid = std::this_thread::get_id();
    if (!Util::ResourceManager::HasCublasHandle(deviceId, tid))
    {
        Util::ResourceManager::AddCublasHandle(deviceId, tid);
    }
    auto* handle = Util::ResourceManager::GetCublasHandle(
        deviceId, tid);
    hipblasSetMathMode(*handle, HIPBLAS_TF32_TENSOR_OP_MATH);

    const float alpha = 1.0f;
    const float beta = 1.0f;

    const auto strideA = M * K;
    const auto strideB = K * N;
    const auto strideOut = M * N;

    const float* ptrA = A;
    const float* ptrB = B;
    const float* ptrC = C;
    float* ptrOut = out;

    Compute::Cuda::CopyDeviceToDevice(ptrOut, ptrC, totalSize * sizeof(float));

    CHECK_CUBLAS(hipblasGemmStridedBatchedEx(
        *handle, HIPBLAS_OP_N, HIPBLAS_OP_N, static_cast<int>(N),
        static_cast<int>(M), static_cast<int>(K), &alpha, ptrB, HIP_R_32F,
        static_cast<int>(N), strideB, ptrA, HIP_R_32F, static_cast<int>(K),
        strideA, &beta, ptrOut, HIP_R_32F, static_cast<int>(N), strideOut,
        static_cast<int>(totalSize / strideOut), HIPBLAS_COMPUTE_32F_FAST_TF32,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP))
}

//! Broadcasts operations matrix-wise
//! while broadcastC is false, broadcastOut must be false
__host__ void GemmMatrixWiseBroadcast(float* out, const float* A,
                                      const float* B, const float* C,
                                      unsigned int M, unsigned int N,
                                      unsigned int K, unsigned int batchSize,
                                      bool broadcastA, bool broadcastB,
                                      bool broadcastC, int deviceId)
{
    const auto tid = std::this_thread::get_id();
    if (!Util::ResourceManager::HasCublasHandle(deviceId, tid))
    {
        Util::ResourceManager::AddCublasHandle(deviceId, tid);
    }
    auto* handle = Util::ResourceManager::GetCublasHandle(deviceId, tid);

    hipblasSetMathMode(*handle, HIPBLAS_TF32_TENSOR_OP_MATH);

    const float alpha = 1.0f;
    const float beta = 1.0f;

    const auto strideA = (broadcastA ? 0 : (M * K));
    const auto strideB = (broadcastB ? 0 : (K * N));
    const auto strideOut = M * N;

    if (broadcastC)
    {
        Compute::Cuda::CopyDeviceToDeviceBroadcast(
            out, C, M * N * batchSize * sizeof(float), M * N * sizeof(float));
    }
    else
        Compute::Cuda::CopyDeviceToDevice(out, C,
                                          M * N * batchSize * sizeof(float));

    CHECK_CUBLAS(hipblasGemmStridedBatchedEx(
        *handle, HIPBLAS_OP_N, HIPBLAS_OP_N, static_cast<int>(N),
        static_cast<int>(M), static_cast<int>(K), &alpha, B, HIP_R_32F,
        static_cast<int>(N), strideB, A, HIP_R_32F, static_cast<int>(K),
        strideA, &beta, out, HIP_R_32F, static_cast<int>(N), strideOut,
        static_cast<int>(batchSize), HIPBLAS_COMPUTE_32F_FAST_TF32,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP))
}

__host__ void GemmNormal(float* out, float* A, float* B, float* C,
                         unsigned int paddedM, unsigned int paddedN,
                         unsigned int paddedK, unsigned int batchSize,
                         bool broadcastA, bool broadcastB, bool broadcastC)
{
    auto* streams =
        static_cast<hipStream_t*>(malloc(sizeof(hipStream_t) * batchSize));
    unsigned int blockSize = paddedM * paddedN / 1024 + 1;

    for (unsigned int batchIdx = 0; batchIdx < batchSize; batchIdx++)
    {
        hipStreamCreate(&streams[batchIdx]);

        float* ptrOut = out + paddedM * paddedN * batchIdx;
        const float* ptrA = A + paddedM * paddedK * (broadcastA ? 0 : batchIdx);
        const float* ptrB = B + paddedK * paddedN * (broadcastB ? 0 : batchIdx);
        const float* ptrC = C + paddedM * paddedN * (broadcastC ? 0 : batchIdx);
        GemmSimple<<<blockSize, 1024, 0, streams[batchIdx]>>>(
            ptrOut, ptrA, ptrB, ptrC, paddedM, paddedN, paddedK);
    }

    for (unsigned int batchIdx = 0; batchIdx < batchSize; batchIdx++)
    {
        hipStreamSynchronize(streams[batchIdx]);
        hipStreamDestroy(streams[batchIdx]);
    }

    free(streams);
}
} // namespace Sapphire::Compute::Dense::Cuda
