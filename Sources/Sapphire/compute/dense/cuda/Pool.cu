// Copyright (c) 2021, Justin Kim

// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include <Sapphire/compute/dense/cuda/Pool.cuh>
#include <Sapphire/util/ResourceManager.hpp>
#include <hipDNN.h>

namespace Sapphire::Compute::Dense::Cuda
{
__host__ void CreateCudnnPool2DMetaData(CudnnPool2DMetaData* metaData,
                                        Shape4D xShape, int windowHeight,
                                        int windowWidth, int strideRow,
                                        int strideCol, int rowPadding,
                                        int columnPadding,
                                        hipdnnPoolingMode_t mode,
                                        hipdnnNanPropagation_t nanPropagation,
                                        int deviceId)
{
    Shape4D outputShape = { 0, 0, 0, 0 };
    hipSetDevice(deviceId);
    checkCuDNN(hipdnnCreatePoolingDescriptor(&metaData->PoolDesc));
    checkCuDNN(hipdnnCreateTensorDescriptor(&metaData->xDesc));
    checkCuDNN(hipdnnCreateTensorDescriptor(&metaData->yDesc));
    checkCuDNN(hipdnnSetTensor4dDescriptor(
        metaData->xDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, xShape.N,
        xShape.Channels, xShape.Height, xShape.Width));
    checkCuDNN(hipdnnSetPooling2dDescriptor(
        metaData->PoolDesc, mode, nanPropagation, windowHeight, windowWidth,
        rowPadding, columnPadding, strideRow, strideCol));
    checkCuDNN(hipdnnGetPooling2dForwardOutputDim(
        metaData->PoolDesc, metaData->xDesc, &outputShape.N,
        &outputShape.Channels, &outputShape.Height, &outputShape.Width));
    checkCuDNN(hipdnnSetTensor4dDescriptor(
        metaData->yDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, outputShape.N,
        outputShape.Channels, outputShape.Height, outputShape.Width));
}

__host__ void CudnnPoolForward2d(CudnnPool2DMetaData* metaData, float* y,
                                 float* x,
                                 float* alpha, float* beta, int deviceId)
{
    hipdnnHandle_t* handle = Util::ResourceManager::GetCudnnHandle(
        deviceId, std::this_thread::get_id());
    hipSetDevice(deviceId);
    checkCuDNN(hipdnnPoolingForward(*handle, metaData->PoolDesc, alpha,
                                   metaData->xDesc, x, beta, metaData->yDesc,
                                   y));
}

__host__ void CudnnPoolBackward2d(CudnnPool2DMetaData* metaData, float* y,
                                  float* dy,
                                  float* x, float* dx, float* alpha,
                                  float* beta, int deviceId)
{
    hipdnnHandle_t* handle = Util::ResourceManager::GetCudnnHandle(
        deviceId, std::this_thread::get_id());
    hipSetDevice(deviceId);
    checkCuDNN(hipdnnPoolingBackward(
        *handle, metaData->PoolDesc, alpha, metaData->yDesc, y,
        metaData->dyDesc, dy, metaData->xDesc, x, beta, metaData->dxDesc, dx));
}

__host__ void Pool2DForward(float* y, float* x, Shape4D xShape,
                            int windowHeight, int windowWidth, int strideRow,
                            int strideCol, int rowPadding, int columnPadding,
                            hipdnnPoolingMode_t mode,
                            hipdnnNanPropagation_t nanPropagation, int deviceId)
{
    const PoolConfig poolConfig = { xShape, windowHeight, windowWidth,
                                    strideRow, strideCol, rowPadding,
                                    columnPadding };
    const auto tid = std::this_thread::get_id();
    if (!Util::ResourceManager::HasCublasHandle(deviceId, tid))
    {
        Util::ResourceManager::AddCublasHandle(deviceId, tid);
    }
    if (!Util::ResourceManager::HasPoolConfig(poolConfig))
    {
        Util::ResourceManager::AddCudnnPool2DMetaData(
            poolConfig, xShape, windowHeight, windowWidth, strideRow,
            strideCol, rowPadding, columnPadding, mode, nanPropagation,
            deviceId);
    }

    float alpha = 1.0f, beta = 0.0f;
    auto* metaData = Util::ResourceManager::GetCudnnPoolMetaData(poolConfig);
    CudnnPoolForward2d(metaData, y, x, &alpha, &beta, deviceId);
}

__host__ void Pool2DBackward(float* y, float* dy, float* x, float* dx,
                             Shape4D xShape,
                             int windowHeight,
                             int windowWidth, int strideRow, int strideCol,
                             int rowPadding,
                             int columnPadding, int deviceId)
{
    const PoolConfig poolConfig = { xShape, windowHeight, windowWidth,
                                    strideRow, strideCol, rowPadding,
                                    columnPadding };

    const auto tid = std::this_thread::get_id();
    if (!Util::ResourceManager::HasCudnnHandle(deviceId, tid))
    {
        throw std::runtime_error(
            "Compute::Dense::Cuda::Pool2DBackward - CudnnHandle was not "
            "found");
    }
    if (!Util::ResourceManager::HasPoolConfig(poolConfig))
    {
        throw std::runtime_error(
            "Compute::Dense::Cuda::Pool2DBackward - CudnnPool2DMetaData was not "
            "found");
    }
    float alpha = 1.0f, beta = 0.0f;
    auto* metaData = Util::ResourceManager::GetCudnnPoolMetaData(poolConfig);
    CudnnPoolBackward2d(metaData, y, dy, x, dx, &alpha, &beta, deviceId);
}
}
