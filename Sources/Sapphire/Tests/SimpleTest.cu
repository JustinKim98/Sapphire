
#include <hip/hip_runtime.h>
// Copyright (c) 2021, Justin Kim

// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#ifdef WITH_CUDA
#include <Sapphire/compute/cudaUtil/CudaParams.cuh>
#include <cuda_runtime.h>
#include <Sapphire/compute/cudaUtil/Memory.hpp>
#include <Sapphire/Tests/CudaFunctionalityTest.cuh>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include "cublas_v2.h"
#define M 6
#define N 5
#define IDX2F(i, j, ld) ((((j)-1) * (ld)) + ((i)-1))

namespace Sapphire::Test
{
int PrintCudaVersion()
{
    int runtime_ver;
    cudaRuntimeGetVersion(&runtime_ver);
    std::cout << "CUDA Runtime version: " << runtime_ver << std::endl;

    int driver_ver;
    cudaDriverGetVersion(&driver_ver);
    std::cout << "CUDA Driver version: " << driver_ver << std::endl;

    return EXIT_SUCCESS;
}

int MallocTest()
{
    float* ptr;

    Compute::Cuda::CudaMalloc((void**)&ptr, 100);
    Compute::Cuda::CudaFree(ptr);

    return EXIT_SUCCESS;
}

static __inline__ void modify(cublasHandle_t handle, float* m, int ldm, int n,
                              int p, int q, float alpha, float beta)
{
    cublasSscal(handle, n - q + 1, &alpha, &m[IDX2F(p, q, ldm)], ldm);
    cublasSscal(handle, ldm - p + 1, &beta, &m[IDX2F(p, q, ldm)], 1);
}

int CublasTest()
{
    cudaError_t cudaStat;
    cublasStatus_t stat;
    cublasHandle_t handle;
    int i, j;
    float* devPtrA;
    float* a = 0;
    a = (float*)malloc(M * N * sizeof(*a));
    if (!a)
    {
        printf("host memory allocation failed");
        return EXIT_FAILURE;
    }
    for (j = 1; j <= N; j++)
    {
        for (i = 1; i <= M; i++)
        {
            a[IDX2F(i, j, M)] = (float)((i - 1) * M + j);
        }
    }
    cudaStat = cudaMalloc((void**)&devPtrA, M * N * sizeof(*a));
    if (cudaStat != cudaSuccess)
    {
        printf("device memory allocation failed");
        return EXIT_FAILURE;
    }
    stat = cublasCreate(&handle);
    if (stat != CUBLAS_STATUS_SUCCESS)
    {
        printf("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    stat = cublasSetMatrix(M, N, sizeof(*a), a, M, devPtrA, M);
    if (stat != CUBLAS_STATUS_SUCCESS)
    {
        printf("data download failed");
        cudaFree(devPtrA);
        cublasDestroy(handle);
        return EXIT_FAILURE;
    }
    modify(handle, devPtrA, M, N, 2, 3, 16.0f, 12.0f);
    stat = cublasGetMatrix(M, N, sizeof(*a), devPtrA, M, a, M);
    if (stat != CUBLAS_STATUS_SUCCESS)
    {
        printf("data upload failed");
        cudaFree(devPtrA);
        cublasDestroy(handle);
        return EXIT_FAILURE;
    }
    cudaFree(devPtrA);
    cublasDestroy(handle);
    for (j = 1; j <= N; j++)
    {
        for (i = 1; i <= M; i++)
        {
            printf("%7.0f", a[IDX2F(i, j, M)]);
        }
        printf("\n");
    }
    free(a);
    return EXIT_SUCCESS;
}
} // namespace Sapphire::Test

#endif
